#ifndef CUDA_VOLUME_DEFORM_SPH_CU
#define CUDA_VOLUME_DEFORM_SPH_CU
//#include "cuda_volume_deform_sph.h"

// Compilation settings

#ifdef USE_DOUBLE_PRECISION
// Types
using PrecisionType = double;
using PrecisionType3 = double3;
// Constants
#define _PI_ (3.1415926535897931e+0)
// Functions
#define SQRT sqrt
#define ATAN2 atan2
#define COS cos
#define SIN sin

#else
// Types
using PrecisionType = float;
using PrecisionType3 = float3;
// Constants
#define _PI_ (3.1415926535897f)
// Functions
#define SQRT sqrtf
#define ATAN2 atan2f
#define COS cosf
#define SIN sinf

#endif// USE_DOUBLE_PRECISION

#ifdef USE_SCATTERED_ZSH_CLNM
using ClnmType = PrecisionType*;
using ZshParamsType =
    struct ZSHparams { int *vL1, *vN, *vL2, *vM; unsigned size; };
#else
using ClnmType = PrecisionType3*;
using ZshParamsType = int4*;
#endif// USE_SCATTERED_ZSH_CLNM

// Compilation settings - end

// Define used data structures

struct ImageData
{
    int xShift;
    int yShift;
    int zShift;

    int xDim;
    int yDim;
    int zDim;

    PrecisionType* data;
};
/*
#ifdef USE_SCATTERED_ZSH_CLNM
struct ZSHparams 
{
    int* vL1;
    int* vN;
    int* vL2;
    int* vM;
    unsigned size;
};
#endif
*/
struct Volumes 
{
    ImageData* I;
    ImageData* R;
    unsigned size;
};

struct IROimages 
{
    ImageData VI;
    ImageData VR;
    ImageData VO;
};

struct DeformImages 
{
    ImageData Gx;
    ImageData Gy;
    ImageData Gz;
};

// CUDA kernel defines
#define BLOCK_X_DIM 8
#define BLOCK_Y_DIM 4
#define BLOCK_Z_DIM 4
#define TOTAL_BLOCK_SIZE (BLOCK_X_DIM * BLOCK_Y_DIM * BLOCK_Z_DIM)

// ImageData macros

// Index to global memory
#define GET_IDX(ImD,k,i,j) \
    ((ImD).xDim * (ImD).yDim * (k) + (ImD).xDim * (i) + (j))

// Logical index = Physical index + shift
#define P2L_X_IDX(ImD,j) \
    ((j) + (ImD).xShift)

#define P2L_Y_IDX(ImD,i) \
    ((i) + (ImD).yShift)

#define P2L_Z_IDX(ImD,k) \
    ((k) + (ImD).zShift)

// Physical index = Logical index - shift
#define L2P_X_IDX(ImD,j) \
    ((j) - (ImD).xShift)

#define L2P_Y_IDX(ImD,i) \
    ((i) - (ImD).yShift)

#define L2P_Z_IDX(ImD,k) \
    ((k) - (ImD).zShift)

// Element access
#define ELEM_3D(ImD,k,i,j) \
    ((ImD).data[GET_IDX((ImD), (k), (i), (j))])

#define ELEM_3D_SHIFTED(ImD,k,i,j) \
    (ELEM_3D((ImD), (k) - (ImD).zShift, (i) - (ImD).yShift, (j) - (ImD).xShift))

// Utility macros
#define MY_OUTSIDE(ImD,k,i,j) \
    ((j) < (ImD).xShift || (j) > (ImD).xShift + (ImD).xDim - 1 || \
     (i) < (ImD).yShift || (i) > (ImD).yShift + (ImD).yDim - 1 || \
     (k) < (ImD).zShift || (k) > (ImD).zShift + (ImD).zDim - 1)

// Smart casting to selected precision (at compile time)
// ...just shorter static_cast
#define CST(num) (static_cast<PrecisionType>((num)))

#define FLOOR(x) (((x) == (int)(x)) ? (int)(x):(((x) > 0) ? (int)(x) : \
                  (int)((x) - 1)))
#define LIN_INTERP(a, l, h) ((l) + ((h) - (l)) * (a))

// Forward declarations
__device__ PrecisionType ZernikeSphericalHarmonics(int l1, int n, int l2, int m,
        PrecisionType xr, PrecisionType yr, PrecisionType zr, PrecisionType r);

__device__ PrecisionType interpolatedElement3D(ImageData ImD,
        PrecisionType x, PrecisionType y, PrecisionType z,
        PrecisionType doutside_value = 0);

/*
 * The beast
 */
extern "C" __global__ void computeDeform(
        PrecisionType Rmax2,
        PrecisionType iRmax,
        IROimages images,
        ZshParamsType zshparams,
        ClnmType clnm,
        int steps,
        Volumes volumes,
        DeformImages deformImages,
        bool applyTransformation,
        bool saveDeformation,
        PrecisionType* g_outArr
        ) 
{
    __shared__ PrecisionType sumArray[TOTAL_BLOCK_SIZE * 4];

    // Compute thread index in a block
    int tIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Get physical indexes
    int kPhys = blockIdx.z * blockDim.z + threadIdx.z;
    int iPhys = blockIdx.y * blockDim.y + threadIdx.y;
    int jPhys = blockIdx.x * blockDim.x + threadIdx.x;

    // Update to logical indexes (calculations expect logical indexing)
    int k = P2L_Z_IDX(images.VR, kPhys);
    int i = P2L_Y_IDX(images.VR, iPhys);
    int j = P2L_X_IDX(images.VR, jPhys);

    PrecisionType r2 = k*k + i*i + j*j;
    PrecisionType rr = SQRT(r2) * iRmax;
    PrecisionType gx = 0.0, gy = 0.0, gz = 0.0;

    if (r2 < Rmax2) {
        for (int idx = 0; idx < steps; idx++) {
#ifdef USE_SCATTERED_ZSH_CLNM
            int l1 = zshparams.vL1[idx];
            int n = zshparams.vN[idx];
            int l2 = zshparams.vL2[idx];
            int m = zshparams.vM[idx];
#else
            int l1 = zshparams[idx].w;
            int n = zshparams[idx].x;
            int l2 = zshparams[idx].y;
            int m = zshparams[idx].z;
#endif
            PrecisionType zsph = ZernikeSphericalHarmonics(l1, n, l2, m,
                    j * iRmax, i * iRmax, k * iRmax, rr);

            if (rr > 0 || l2 == 0) {
#ifdef USE_SCATTERED_ZSH_CLNM
                gx += zsph * clnm[idx];
                gy += zsph * clnm[idx + zshparams.size];
                gz += zsph * clnm[idx + zshparams.size * 2];
#else
                gx += zsph * clnm[idx].x;
                gy += zsph * clnm[idx].y;
                gz += zsph * clnm[idx].z;
#endif
            }
        }
    }

    PrecisionType voxelI, voxelR;
    PrecisionType diff;

    PrecisionType localDiff2 = 0.0, localSumVD = 0.0, localModg = 0.0, localNcount = 0.0;

    if (applyTransformation) {
        // Indexing requires physical indexes
        voxelR = ELEM_3D(images.VR, kPhys, iPhys, jPhys);
        // Logical indexes used to check whether the point is in the matrix
        voxelI = interpolatedElement3D(images.VI, j + gx, i + gy, k + gz);

        if (voxelI >= 0.0)
            localSumVD += voxelI;

        ELEM_3D(images.VO, kPhys, iPhys, jPhys) = voxelI;
        diff = voxelR - voxelI;
        localDiff2 += diff * diff;
        localModg += gx*gx + gy*gy + gz*gz;
        localNcount++;
    }

    for (unsigned idv = 0; idv < volumes.size; idv++) {
        voxelR = ELEM_3D(volumes.R[idv], kPhys, iPhys, jPhys);
        voxelI = interpolatedElement3D(volumes.I[idv], j + gx, i + gy, k + gz);

        if (voxelI >= 0.0)
            localSumVD += voxelI;

        diff = voxelR - voxelI;
        localDiff2 += diff * diff;
        localModg += gx*gx + gy*gy + gz*gz;
        localNcount++;
    }

    sumArray[tIdx] = localDiff2;
    sumArray[tIdx + TOTAL_BLOCK_SIZE] = localSumVD;
    sumArray[tIdx + TOTAL_BLOCK_SIZE * 2] = localModg;
    sumArray[tIdx + TOTAL_BLOCK_SIZE * 3] = localNcount;

    __syncthreads();
    
    // Block reduction   
    for (int s = TOTAL_BLOCK_SIZE / 2; s > 0; s /= 2) {
        if (tIdx < s) {
            sumArray[tIdx] += sumArray[tIdx + s];
            sumArray[tIdx + TOTAL_BLOCK_SIZE] += sumArray[tIdx + TOTAL_BLOCK_SIZE + s];
            sumArray[tIdx + TOTAL_BLOCK_SIZE * 2] += sumArray[tIdx + TOTAL_BLOCK_SIZE * 2 + s];
            sumArray[tIdx + TOTAL_BLOCK_SIZE * 3] += sumArray[tIdx + TOTAL_BLOCK_SIZE * 3 + s];
        }
        __syncthreads();
    }

    // Save values to the global memory for later
    if (tIdx == 0) {
        int bIdx = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
        int TOTAL_GRID_SIZE = gridDim.x * gridDim.y * gridDim.z;
        g_outArr[bIdx] = sumArray[0];
        g_outArr[bIdx + TOTAL_GRID_SIZE] = sumArray[TOTAL_BLOCK_SIZE];
        g_outArr[bIdx + TOTAL_GRID_SIZE * 2] = sumArray[TOTAL_BLOCK_SIZE * 2];
        g_outArr[bIdx + TOTAL_GRID_SIZE * 3] = sumArray[TOTAL_BLOCK_SIZE * 3];
    }

    if (saveDeformation) {
        ELEM_3D(deformImages.Gx, kPhys, iPhys, jPhys) = gx;
        ELEM_3D(deformImages.Gy, kPhys, iPhys, jPhys) = gy;
        ELEM_3D(deformImages.Gz, kPhys, iPhys, jPhys) = gz;
    }
}

/*
 * Linear interpolation
 */
__device__ PrecisionType interpolatedElement3D(ImageData ImD,
        PrecisionType x, PrecisionType y, PrecisionType z,
        PrecisionType outside_value) 
{
        int x0 = FLOOR(x);
        PrecisionType fx = x - x0;
        int x1 = x0 + 1;

        int y0 = FLOOR(y);
        PrecisionType fy = y - y0;
        int y1 = y0 + 1;

        int z0 = FLOOR(z);
        PrecisionType fz = z - z0;
        int z1 = z0 + 1;

        PrecisionType d000 = (MY_OUTSIDE(ImD, z0, y0, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z0, y0, x0);
        PrecisionType d001 = (MY_OUTSIDE(ImD, z0, y0, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z0, y0, x1);
        PrecisionType d010 = (MY_OUTSIDE(ImD, z0, y1, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z0, y1, x0);
        PrecisionType d011 = (MY_OUTSIDE(ImD, z0, y1, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z0, y1, x1);
        PrecisionType d100 = (MY_OUTSIDE(ImD, z1, y0, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z1, y0, x0);
        PrecisionType d101 = (MY_OUTSIDE(ImD, z1, y0, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z1, y0, x1);
        PrecisionType d110 = (MY_OUTSIDE(ImD, z1, y1, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z1, y1, x0);
        PrecisionType d111 = (MY_OUTSIDE(ImD, z1, y1, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, z1, y1, x1);

        PrecisionType dx00 = LIN_INTERP(fx, d000, d001);
        PrecisionType dx01 = LIN_INTERP(fx, d100, d101);
        PrecisionType dx10 = LIN_INTERP(fx, d010, d011);
        PrecisionType dx11 = LIN_INTERP(fx, d110, d111);
        PrecisionType dxy0 = LIN_INTERP(fy, dx00, dx10);
        PrecisionType dxy1 = LIN_INTERP(fy, dx01, dx11);

        return LIN_INTERP(fz, dxy0, dxy1);
}

/*
 * ZSH
 */
__device__ PrecisionType ZernikeSphericalHarmonics(int l1, int n, int l2, int m, PrecisionType xr, PrecisionType yr, PrecisionType zr, PrecisionType r)
{
	// General variables
	PrecisionType r2=r*r,xr2=xr*xr,yr2=yr*yr,zr2=zr*zr;

	//Variables needed for l>=5
	PrecisionType tht=CST(0.0),phi=CST(0.0),cost=CST(0.0),sint=CST(0.0),cost2=CST(0.0),sint2=CST(0.0);
	if (l2>=5)
	{
		tht = ATAN2(yr,xr);
		phi = ATAN2(zr,SQRT(xr2 + yr2));
		sint = SIN(phi); cost = COS(tht);
		sint2 = sint*sint; cost2 = cost*cost;
	}

	// Zernike polynomial
	PrecisionType R=CST(0.0);

	switch (l1)
	{
	case 0:
		R = SQRT(CST(3));
		break;
	case 1:
		R = SQRT(CST(5))*r;
		break;
	case 2:
		switch (n)
		{
		case 0:
			R = CST(-0.5)*SQRT(CST(7))*(CST(2.5)*(1-2*r2)+CST(0.5));
			break;
		case 2:
			R = SQRT(CST(7))*r2;
			break;
		} break;
	case 3:
		switch (n)
		{
		case 1:
			R = CST(-1.5)*r*(CST(3.5)*(1-2*r2)+CST(1.5));
			break;
		case 3:
			R = 3*r2*r;
		} break;
	case 4:
		switch (n)
		{
		case 0:
			R = SQRT(CST(11))*((63*r2*r2/8)-(35*r2/4)+(CST(15)/CST(8)));
			break;
		case 2:
			R = CST(-0.5)*SQRT(CST(11))*r2*(CST(4.5)*(1-2*r2)+CST(2.5));
			break;
		case 4:
			R = SQRT(CST(11))*r2*r2;
			break;
		} break;
	case 5:
		switch (n)
		{
		case 1:
			R = SQRT(CST(13))*r*((99*r2*r2/8)-(63*r2/4)+(CST(35)/CST(8)));
			break;
		case 3:
			R = CST(-0.5)*SQRT(CST(13))*r2*r*(CST(5.5)*(1-2*r2)+CST(3.5));
			break;
		} break;
	}

	// Spherical harmonic
	PrecisionType Y=CST(0.0);

	switch (l2)
	{
	case 0:
		Y = (CST(1.0)/CST(2.0))*SQRT((PrecisionType) CST(1.0)/_PI_);
		break;
	case 1:
		switch (m)
		{
		case -1:
			Y = SQRT(CST(3.0)/(CST(4.0)*_PI_))*yr;
			break;
		case 0:
			Y = SQRT(CST(3.0)/(CST(4.0)*_PI_))*zr;
			break;
		case 1:
			Y = SQRT(CST(3.0)/(CST(4.0)*_PI_))*xr;
			break;
		} break;
	case 2:
		switch (m)
		{
		case -2:
			Y = SQRT(CST(15.0)/(CST(4.0)*_PI_))*xr*yr;
			break;
		case -1:
			Y = SQRT(CST(15.0)/(CST(4.0)*_PI_))*zr*yr;
			break;
		case 0:
			Y = SQRT(CST(5.0)/(CST(16.0)*_PI_))*(-xr2-yr2+CST(2.0)*zr2);
			break;
		case 1:
			Y = SQRT(CST(15.0)/(CST(4.0)*_PI_))*xr*zr;
			break;
		case 2:
			Y = SQRT(CST(15.0)/(CST(16.0)*_PI_))*(xr2-yr2);
			break;
		} break;
	case 3:
		switch (m)
		{
		case -3:
			Y = SQRT(CST(35.0)/(CST(16.0)*CST(2.0)*_PI_))*yr*(CST(3.0)*xr2-yr2);
			break;
		case -2:
			Y = SQRT(CST(105.0)/(CST(4.0)*_PI_))*zr*yr*xr;
			break;
		case -1:
			Y = SQRT(CST(21.0)/(CST(16.0)*CST(2.0)*_PI_))*yr*(CST(4.0)*zr2-xr2-yr2);
			break;
		case 0:
			Y = SQRT(CST(7.0)/(CST(16.0)*_PI_))*zr*(CST(2.0)*zr2-CST(3.0)*xr2-CST(3.0)*yr2);
			break;
		case 1:
			Y = SQRT(CST(21.0)/(CST(16.0)*CST(2.0)*_PI_))*xr*(CST(4.0)*zr2-xr2-yr2);
			break;
		case 2:
			Y = SQRT(CST(105.0)/(CST(16.0)*_PI_))*zr*(xr2-yr2);
			break;
		case 3:
			Y = SQRT(CST(35.0)/(CST(16.0)*CST(2.0)*_PI_))*xr*(xr2-CST(3.0)*yr2);
			break;
		} break;
	case 4:
		switch (m)
		{
		case -4:
			Y = SQRT((CST(35.0)*CST(9.0))/(CST(16.0)*_PI_))*yr*xr*(xr2-yr2);
			break;
		case -3:
			Y = SQRT((CST(9.0)*CST(35.0))/(CST(16.0)*CST(2.0)*_PI_))*yr*zr*(CST(3.0)*xr2-yr2);
			break;
		case -2:
			Y = SQRT((CST(9.0)*CST(5.0))/(CST(16.0)*_PI_))*yr*xr*(CST(7.0)*zr2-(xr2+yr2+zr2));
			break;
		case -1:
			Y = SQRT((CST(9.0)*CST(5.0))/(CST(16.0)*CST(2.0)*_PI_))*yr*zr*(CST(7.0)*zr2-CST(3.0)*(xr2+yr2+zr2));
			break;
		case 0:
			Y = SQRT(CST(9.0)/(CST(16.0)*CST(16.0)*_PI_))*(CST(35.0)*zr2*zr2-CST(30.0)*zr2+CST(3.0));
			break;
		case 1:
			Y = SQRT((CST(9.0)*CST(5.0))/(CST(16.0)*CST(2.0)*_PI_))*xr*zr*(CST(7.0)*zr2-CST(3.0)*(xr2+yr2+zr2));
			break;
		case 2:
			Y = SQRT((CST(9.0)*CST(5.0))/(CST(8.0)*CST(8.0)*_PI_))*(xr2-yr2)*(CST(7.0)*zr2-(xr2+yr2+zr2));
			break;
		case 3:
			Y = SQRT((CST(9.0)*CST(35.0))/(CST(16.0)*CST(2.0)*_PI_))*xr*zr*(xr2-CST(3.0)*yr2);
			break;
		case 4:
			Y = SQRT((CST(9.0)*CST(35.0))/(CST(16.0)*CST(16.0)*_PI_))*(xr2*(xr2-CST(3.0)*yr2)-yr2*(CST(3.0)*xr2-yr2));
			break;
		} break;
	case 5:
		switch (m)
		{
		case -5:
			Y = (CST(3.0)/CST(16.0))*SQRT(CST(77.0)/(CST(2.0)*_PI_))*sint2*sint2*sint*SIN(CST(5.0)*phi);
			break;
		case -4:
			Y = (CST(3.0)/CST(8.0))*SQRT(CST(385.0)/(CST(2.0)*_PI_))*sint2*sint2*SIN(CST(4.0)*phi);
			break;
		case -3:
			Y = (CST(1.0)/CST(16.0))*SQRT(CST(385.0)/(CST(2.0)*_PI_))*sint2*sint*(CST(9.0)*cost2-CST(1.0))*SIN(CST(3.0)*phi);
			break;
		case -2:
			Y = (CST(1.0)/CST(4.0))*SQRT(CST(1155.0)/(CST(4.0)*_PI_))*sint2*(CST(3.0)*cost2*cost-cost)*SIN(CST(2.0)*phi);
			break;
		case -1:
			Y = (CST(1.0)/CST(8.0))*SQRT(CST(165.0)/(CST(4.0)*_PI_))*sint*(CST(21.0)*cost2*cost2-CST(14.0)*cost2+1)*SIN(phi);
			break;
		case 0:
			Y = (CST(1.0)/CST(16.0))*SQRT(CST(11.0)/_PI_)*(CST(63.0)*cost2*cost2*cost-CST(70.0)*cost2*cost+CST(15.0)*cost);
			break;
		case 1:
			Y = (CST(1.0)/CST(8.0))*SQRT(CST(165.0)/(CST(4.0)*_PI_))*sint*(CST(21.0)*cost2*cost2-CST(14.0)*cost2+1)*COS(phi);
			break;
		case 2:
			Y = (CST(1.0)/CST(4.0))*SQRT(CST(1155.0)/(CST(4.0)*_PI_))*sint2*(CST(3.0)*cost2*cost-cost)*COS(CST(2.0)*phi);
			break;
		case 3:
			Y = (CST(1.0)/CST(16.0))*SQRT(CST(385.0)/(CST(2.0)*_PI_))*sint2*sint*(CST(9.0)*cost2-CST(1.0))*COS(CST(3.0)*phi);
			break;
		case 4:
			Y = (CST(3.0)/CST(8.0))*SQRT(CST(385.0)/(CST(2.0)*_PI_))*sint2*sint2*COS(CST(4.0)*phi);
			break;
		case 5:
			Y = (CST(3.0)/CST(16.0))*SQRT(CST(77.0)/(CST(2.0)*_PI_))*sint2*sint2*sint*COS(CST(5.0)*phi);
			break;
		}break;
	}

	return R*Y;
}


// Function redefinition
#ifdef USE_DOUBLE_PRECISION 
/*
__device__ double ZernikeSphericalHarmonics(int l1, int n, int l2, int m, double xr, double yr, double zr, double r)
{
	// General variables
	double r2=r*r,xr2=xr*xr,yr2=yr*yr,zr2=zr*zr;

	//Variables needed for l>=5
	double tht=0.0,phi=0.0,cost=0.0,sint=0.0,cost2=0.0,sint2=0.0;
	if (l2>=5)
	{
		tht = atan2(yr,xr);
		phi = atan2(zr,sqrt(xr2 + yr2));
		sint = sin(phi); cost = cos(tht);
		sint2 = sint*sint; cost2 = cost*cost;
	}

	// Zernike polynomial
	double R=0.0;

	switch (l1)
	{
	case 0:
		R = sqrt((double) 3);
		break;
	case 1:
		R = sqrt((double) 5)*r;
		break;
	case 2:
		switch (n)
		{
		case 0:
			R = -0.5*sqrt((double) 7)*(2.5*(1-2*r2)+0.5);
			break;
		case 2:
			R = sqrt((double) 7)*r2;
			break;
		} break;
	case 3:
		switch (n)
		{
		case 1:
			R = -1.5*r*(3.5*(1-2*r2)+1.5);
			break;
		case 3:
			R = 3*r2*r;
		} break;
	case 4:
		switch (n)
		{
		case 0:
			R = sqrt((double) 11)*((63*r2*r2/8)-(35*r2/4)+(15/8));
			break;
		case 2:
			R = -0.5*sqrt((double) 11)*r2*(4.5*(1-2*r2)+2.5);
			break;
		case 4:
			R = sqrt((double) 11)*r2*r2;
			break;
		} break;
	case 5:
		switch (n)
		{
		case 1:
			R = sqrt((double) 13)*r*((99*r2*r2/8)-(63*r2/4)+(35/8));
			break;
		case 3:
			R = -0.5*sqrt((double) 13)*r2*r*(5.5*(1-2*r2)+3.5);
			break;
		} break;
	}

	// Spherical harmonic
	double Y=0.0;

	switch (l2)
	{
	case 0:
		Y = (1.0/2.0)*sqrt((double) 1.0/_PI_);
		break;
	case 1:
		switch (m)
		{
		case -1:
			Y = sqrt(3.0/(4.0*_PI_))*yr;
			break;
		case 0:
			Y = sqrt(3.0/(4.0*_PI_))*zr;
			break;
		case 1:
			Y = sqrt(3.0/(4.0*_PI_))*xr;
			break;
		} break;
	case 2:
		switch (m)
		{
		case -2:
			Y = sqrt(15.0/(4.0*_PI_))*xr*yr;
			break;
		case -1:
			Y = sqrt(15.0/(4.0*_PI_))*zr*yr;
			break;
		case 0:
			Y = sqrt(5.0/(16.0*_PI_))*(-xr2-yr2+2.0*zr2);
			break;
		case 1:
			Y = sqrt(15.0/(4.0*_PI_))*xr*zr;
			break;
		case 2:
			Y = sqrt(15.0/(16.0*_PI_))*(xr2-yr2);
			break;
		} break;
	case 3:
		switch (m)
		{
		case -3:
			Y = sqrt(35.0/(16.0*2.0*_PI_))*yr*(3.0*xr2-yr2);
			break;
		case -2:
			Y = sqrt(105.0/(4.0*_PI_))*zr*yr*xr;
			break;
		case -1:
			Y = sqrt(21.0/(16.0*2.0*_PI_))*yr*(4.0*zr2-xr2-yr2);
			break;
		case 0:
			Y = sqrt(7.0/(16.0*_PI_))*zr*(2.0*zr2-3.0*xr2-3.0*yr2);
			break;
		case 1:
			Y = sqrt(21.0/(16.0*2.0*_PI_))*xr*(4.0*zr2-xr2-yr2);
			break;
		case 2:
			Y = sqrt(105.0/(16.0*_PI_))*zr*(xr2-yr2);
			break;
		case 3:
			Y = sqrt(35.0/(16.0*2.0*_PI_))*xr*(xr2-3.0*yr2);
			break;
		} break;
	case 4:
		switch (m)
		{
		case -4:
			Y = sqrt((35.0*9.0)/(16.0*_PI_))*yr*xr*(xr2-yr2);
			break;
		case -3:
			Y = sqrt((9.0*35.0)/(16.0*2.0*_PI_))*yr*zr*(3.0*xr2-yr2);
			break;
		case -2:
			Y = sqrt((9.0*5.0)/(16.0*_PI_))*yr*xr*(7.0*zr2-(xr2+yr2+zr2));
			break;
		case -1:
			Y = sqrt((9.0*5.0)/(16.0*2.0*_PI_))*yr*zr*(7.0*zr2-3.0*(xr2+yr2+zr2));
			break;
		case 0:
			Y = sqrt(9.0/(16.0*16.0*_PI_))*(35.0*zr2*zr2-30.0*zr2+3.0);
			break;
		case 1:
			Y = sqrt((9.0*5.0)/(16.0*2.0*_PI_))*xr*zr*(7.0*zr2-3.0*(xr2+yr2+zr2));
			break;
		case 2:
			Y = sqrt((9.0*5.0)/(8.0*8.0*_PI_))*(xr2-yr2)*(7.0*zr2-(xr2+yr2+zr2));
			break;
		case 3:
			Y = sqrt((9.0*35.0)/(16.0*2.0*_PI_))*xr*zr*(xr2-3.0*yr2);
			break;
		case 4:
			Y = sqrt((9.0*35.0)/(16.0*16.0*_PI_))*(xr2*(xr2-3.0*yr2)-yr2*(3.0*xr2-yr2));
			break;
		} break;
	case 5:
		switch (m)
		{
		case -5:
			Y = (3.0/16.0)*sqrt(77.0/(2.0*_PI_))*sint2*sint2*sint*sin(5.0*phi);
			break;
		case -4:
			Y = (3.0/8.0)*sqrt(385.0/(2.0*_PI_))*sint2*sint2*sin(4.0*phi);
			break;
		case -3:
			Y = (1.0/16.0)*sqrt(385.0/(2.0*_PI_))*sint2*sint*(9.0*cost2-1.0)*sin(3.0*phi);
			break;
		case -2:
			Y = (1.0/4.0)*sqrt(1155.0/(4.0*_PI_))*sint2*(3.0*cost2*cost-cost)*sin(2.0*phi);
			break;
		case -1:
			Y = (1.0/8.0)*sqrt(165.0/(4.0*_PI_))*sint*(21.0*cost2*cost2-14.0*cost2+1)*sin(phi);
			break;
		case 0:
			Y = (1.0/16.0)*sqrt(11.0/_PI_)*(63.0*cost2*cost2*cost-70.0*cost2*cost+15.0*cost);
			break;
		case 1:
			Y = (1.0/8.0)*sqrt(165.0/(4.0*_PI_))*sint*(21.0*cost2*cost2-14.0*cost2+1)*cos(phi);
			break;
		case 2:
			Y = (1.0/4.0)*sqrt(1155.0/(4.0*_PI_))*sint2*(3.0*cost2*cost-cost)*cos(2.0*phi);
			break;
		case 3:
			Y = (1.0/16.0)*sqrt(385.0/(2.0*_PI_))*sint2*sint*(9.0*cost2-1.0)*cos(3.0*phi);
			break;
		case 4:
			Y = (3.0/8.0)*sqrt(385.0/(2.0*_PI_))*sint2*sint2*cos(4.0*phi);
			break;
		case 5:
			Y = (3.0/16.0)*sqrt(77.0/(2.0*_PI_))*sint2*sint2*sint*cos(5.0*phi);
			break;
		}break;
	}

	return R*Y;
}
*/
#else
/*
__device__ float ZernikeSphericalHarmonics(int l1, int n, int l2, int m, float xr, float yr, float zr, float r)
{
	// General variables
	float r2=r*r,xr2=xr*xr,yr2=yr*yr,zr2=zr*zr;

	//Variables needed for l>=5
	float tht=0.0f,phi=0.0f,cost=0.0f,sint=0.0f,cost2=0.0f,sint2=0.0f;
	if (l2>=5)
	{
		tht = atan2f(yr,xr);
		phi = atan2f(zr,sqrtf(xr2 + yr2));
		sint = sinf(phi); cost = cosf(tht);
		sint2 = sint*sint; cost2 = cost*cost;
	}

	// Zernike polynomial
	float R=0.0f;

	switch (l1)
	{
	case 0:
		R = sqrtf((float) 3);
		break;
	case 1:
		R = sqrtf((float) 5)*r;
		break;
	case 2:
		switch (n)
		{
		case 0:
			R = -0.5f*sqrtf((float) 7)*(2.5f*(1-2*r2)+0.5f);
			break;
		case 2:
			R = sqrtf((float) 7)*r2;
			break;
		} break;
	case 3:
		switch (n)
		{
		case 1:
			R = -1.5f*r*(3.5f*(1-2*r2)+1.5f);
			break;
		case 3:
			R = 3*r2*r;
		} break;
	case 4:
		switch (n)
		{
		case 0:
			R = sqrtf((float) 11)*((63*r2*r2/8)-(35*r2/4)+(15/8));
			break;
		case 2:
			R = -0.5f*sqrtf((float) 11)*r2*(4.5f*(1-2*r2)+2.5f);
			break;
		case 4:
			R = sqrtf((float) 11)*r2*r2;
			break;
		} break;
	case 5:
		switch (n)
		{
		case 1:
			R = sqrtf((float) 13)*r*((99*r2*r2/8)-(63*r2/4)+(35/8));
			break;
		case 3:
			R = -0.5f*sqrtf((float) 13)*r2*r*(5.5f*(1-2*r2)+3.5f);
			break;
		} break;
	}

	// Spherical harmonic
	float Y=0.0f;

	switch (l2)
	{
	case 0:
		Y = (1.0f/2.0f)*sqrtf((float) 1.0f/_PI_);
		break;
	case 1:
		switch (m)
		{
		case -1:
			Y = sqrtf(3.0f/(4.0f*_PI_))*yr;
			break;
		case 0:
			Y = sqrtf(3.0f/(4.0f*_PI_))*zr;
			break;
		case 1:
			Y = sqrtf(3.0f/(4.0f*_PI_))*xr;
			break;
		} break;
	case 2:
		switch (m)
		{
		case -2:
			Y = sqrtf(15.0f/(4.0f*_PI_))*xr*yr;
			break;
		case -1:
			Y = sqrtf(15.0f/(4.0f*_PI_))*zr*yr;
			break;
		case 0:
			Y = sqrtf(5.0f/(16.0f*_PI_))*(-xr2-yr2+2.0f*zr2);
			break;
		case 1:
			Y = sqrtf(15.0f/(4.0f*_PI_))*xr*zr;
			break;
		case 2:
			Y = sqrtf(15.0f/(16.0f*_PI_))*(xr2-yr2);
			break;
		} break;
	case 3:
		switch (m)
		{
		case -3:
			Y = sqrtf(35.0f/(16.0f*2.0f*_PI_))*yr*(3.0f*xr2-yr2);
			break;
		case -2:
			Y = sqrtf(105.0f/(4.0f*_PI_))*zr*yr*xr;
			break;
		case -1:
			Y = sqrtf(21.0f/(16.0f*2.0f*_PI_))*yr*(4.0f*zr2-xr2-yr2);
			break;
		case 0:
			Y = sqrtf(7.0f/(16.0f*_PI_))*zr*(2.0f*zr2-3.0f*xr2-3.0f*yr2);
			break;
		case 1:
			Y = sqrtf(21.0f/(16.0f*2.0f*_PI_))*xr*(4.0f*zr2-xr2-yr2);
			break;
		case 2:
			Y = sqrtf(105.0f/(16.0f*_PI_))*zr*(xr2-yr2);
			break;
		case 3:
			Y = sqrtf(35.0f/(16.0f*2.0f*_PI_))*xr*(xr2-3.0f*yr2);
			break;
		} break;
	case 4:
		switch (m)
		{
		case -4:
			Y = sqrtf((35.0f*9.0f)/(16.0f*_PI_))*yr*xr*(xr2-yr2);
			break;
		case -3:
			Y = sqrtf((9.0f*35.0f)/(16.0f*2.0f*_PI_))*yr*zr*(3.0f*xr2-yr2);
			break;
		case -2:
			Y = sqrtf((9.0f*5.0f)/(16.0f*_PI_))*yr*xr*(7.0f*zr2-(xr2+yr2+zr2));
			break;
		case -1:
			Y = sqrtf((9.0f*5.0f)/(16.0f*2.0f*_PI_))*yr*zr*(7.0f*zr2-3.0f*(xr2+yr2+zr2));
			break;
		case 0:
			Y = sqrtf(9.0f/(16.0f*16.0f*_PI_))*(35.0f*zr2*zr2-30.0f*zr2+3.0f);
			break;
		case 1:
			Y = sqrtf((9.0f*5.0f)/(16.0f*2.0f*_PI_))*xr*zr*(7.0f*zr2-3.0f*(xr2+yr2+zr2));
			break;
		case 2:
			Y = sqrtf((9.0f*5.0f)/(8.0f*8.0f*_PI_))*(xr2-yr2)*(7.0f*zr2-(xr2+yr2+zr2));
			break;
		case 3:
			Y = sqrtf((9.0f*35.0f)/(16.0f*2.0f*_PI_))*xr*zr*(xr2-3.0f*yr2);
			break;
		case 4:
			Y = sqrtf((9.0f*35.0f)/(16.0f*16.0f*_PI_))*(xr2*(xr2-3.0f*yr2)-yr2*(3.0f*xr2-yr2));
			break;
		} break;
	case 5:
		switch (m)
		{
		case -5:
			Y = (3.0f/16.0f)*sqrtf(77.0f/(2.0f*_PI_))*sint2*sint2*sint*sinf(5.0f*phi);
			break;
		case -4:
			Y = (3.0f/8.0f)*sqrtf(385.0f/(2.0f*_PI_))*sint2*sint2*sinf(4.0f*phi);
			break;
		case -3:
			Y = (1.0f/16.0f)*sqrtf(385.0f/(2.0f*_PI_))*sint2*sint*(9.0f*cost2-1.0f)*sinf(3.0f*phi);
			break;
		case -2:
			Y = (1.0f/4.0f)*sqrtf(1155.0f/(4.0f*_PI_))*sint2*(3.0f*cost2*cost-cost)*sinf(2.0f*phi);
			break;
		case -1:
			Y = (1.0f/8.0f)*sqrtf(165.0f/(4.0f*_PI_))*sint*(21.0f*cost2*cost2-14.0f*cost2+1)*sinf(phi);
			break;
		case 0:
			Y = (1.0f/16.0f)*sqrtf(11.0f/_PI_)*(63.0f*cost2*cost2*cost-70.0f*cost2*cost+15.0f*cost);
			break;
		case 1:
			Y = (1.0f/8.0f)*sqrtf(165.0f/(4.0f*_PI_))*sint*(21.0f*cost2*cost2-14.0f*cost2+1)*cosf(phi);
			break;
		case 2:
			Y = (1.0f/4.0f)*sqrtf(1155.0f/(4.0f*_PI_))*sint2*(3.0f*cost2*cost-cost)*cosf(2.0f*phi);
			break;
		case 3:
			Y = (1.0f/16.0f)*sqrtf(385.0f/(2.0f*_PI_))*sint2*sint*(9.0f*cost2-1.0f)*cosf(3.0f*phi);
			break;
		case 4:
			Y = (3.0f/8.0f)*sqrtf(385.0f/(2.0f*_PI_))*sint2*sint2*cosf(4.0f*phi);
			break;
		case 5:
			Y = (3.0f/16.0f)*sqrtf(77.0f/(2.0f*_PI_))*sint2*sint2*sint*cosf(5.0f*phi);
			break;
		}break;
	}

	return R*Y;
}
*/
#endif//COMP_DOUBLE


#endif //CUDA_VOLUME_DEFORM_SPH_CU
